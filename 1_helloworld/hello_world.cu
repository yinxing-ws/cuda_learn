
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_world(void)
{
    printf("GPU: Hello world\n");
}
int main(int argc, char **argv) 
{
    printf("CPU: hello world\n");
    hello_world<<<1,10>>>();
    hipDeviceReset();
    return 0;
}